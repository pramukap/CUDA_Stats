#include "hip/hip_runtime.h"
#include    "hip/hip_runtime.h"
#include    "vec_kernels.cuh"

#include    <cstddef>

__device__
void    vec_dot_product(double *a, double *b, double *out, size_t stride, size_t n)
{
    extern __shared__ double temp[];
    size_t tid = threadIdx.x;
    size_t gid = blockIdx.x * blockDim.x + tid;
    size_t idx = gid * stride;

    temp[tid] = (idx < n) ? a[idx] * b[idx] : 0;
    __syncthreads();

    for (size_t shf = blockDim.x / 2; shf > 0; shf >>= 1) {
        if (tid < shf) {
            temp[tid] += temp[tid + shf];  
        }

        __syncthreads();
    }

    if (tid == 0)
        out[blockIdx.x] = temp[0];
}

__device__
void    vec_scalar_mul(double *a, double *out, double c, size_t stride, size_t n) 
{
    size_t tid = threadIdx.x;
    size_t gid = blockIdx.x * blockDim.x + tid;
    size_t idx = gid * stride;

    if (idx < n)
        out[idx] = a[idx] * c;
}
