#include "kmeansHelper.cu"
//#include "University_Data.h"
#include "Iris_Data.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

// Assume data is filled
// Assume centroids is allocated
void kmeans(double* data, int m, int n, int k, double* centroids, int iterations){

    double *data_d;
    double *centroids_d;
    int *counts;
    int *labels;
    double *distances;

    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&counts, k*sizeof(int));
    hipMalloc((void**)&labels, m*sizeof(int));
    /* old distance
    hipMalloc((void**)&distances, k*sizeof(double));
    */
    hipMalloc((void**)&distances, m*sizeof(double));

    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);

    // Initalize centroids using random partition of data into k groups
    init_labels<<<m, 1>>>(labels, k);
    init_zeros<<<k, 1>>>(counts);
    init_zeros<<<k, n>>>(centroids_d);

    findNewCentroids<<<m, n>>>(data_d, centroids_d, labels, m, n, k, counts);
    divide_by_count<<<k, n>>>(centroids_d, counts, n, k);

    // Set number of iterations
    for(int step__ = 0; step__ < iterations; step__++){

        // Assignment Step
        init_zeros<<<m, 1>>>(distances);
        assignClasses<<<m, 1>>>(data_d, centroids_d, m, n, k, labels, distances);


        // OLD ASSIGNMENT
        /*
        for(int point = 0; point < m; point++){

            subtractPointFromMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);
            getDistances<<<k, 1>>>(centroids_d, distances, k, n);
            assignClass<<<1, 1>>>(distances, labels, k, point);
            addPointToMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);

        }
        */

        // Update Means Step
        init_zeros<<<k, 1>>>(counts);
        init_zeros<<<k, n>>>(centroids_d);
        findNewCentroids<<<m, n>>>(data_d, centroids_d, labels, m, n, k, counts);
        divide_by_count<<<k, n>>>(centroids_d, counts, n, k);

    }
    hipMemcpy(centroids, centroids_d, k*n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(centroids_d);
    hipFree(counts);
    hipFree(labels);
    hipFree(distances);

}

// Assume centroids is filled
// Assume labels is allocated
// Assume data is filled
void kmeans_classify(double * centroids, double * data, int *labels_h, int m, int n, int k){

    double *data_d;
    double *centroids_d;
    int *labels;
    double *distances;

    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&labels, m*sizeof(int));
    hipMalloc((void**)&distances, k*sizeof(double));

    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(centroids_d, centroids, k*n*sizeof(double), hipMemcpyHostToDevice);

    for(int point = 0; point < m; point++){
        subtractPointFromMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);
        getDistances<<<k, 1>>>(centroids_d, distances, k, n);
        assignClass<<<1, 1>>>(distances, labels, k, point);
        addPointToMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);
    }

    hipMemcpy(labels_h, labels, m*sizeof(int), hipMemcpyDeviceToHost);
}

void run_small_kmeans_test(){

    int m = 12;
    int n = 2;
    int k = 3;
    int iterations = 100;

    double data[m*n] = {0, 1, 1, 0, 1, 1,0,0, 5, 6, 6, 7,7,5 ,5, 5, 0, 8, 1, 9, 0, 9,1,8};
    double* centroids = (double*) malloc(k*n*sizeof(double));

    kmeans(data, m, n, k, centroids, iterations);

}

void printConfusionMatrix(int *actual, int*expect){

    int tp = 0;
    int fp = 0;
    int fn = 0;
    int tn = 0;

    for(int i = 0; i < 777; i++){
        if(actual[i] == expect[i]){
            if(actual[i] == 1){
                tp++;
            } else {
                tn++;
            }
        } else {
            if (actual[i] == 1){
                fp++;
            }else{
                fn++;
            }
        }
    }
    printf("\n");
    printf("TP: %d\nFP: %d\nFN: %d\nTN: %d\n", tp, fp, fn, tn);


}
/*
void run_uni_data_test(){

    int m = 777;
    int n = 17;
    int k = 2;
    int iterations = 10;

    double* centroids = (double*) malloc(k*n*sizeof(double));
    kmeans(data, m, n, k, centroids, iterations);
    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }

    int *labels = (int *) malloc(sizeof(int) * m);
    kmeans_classify(centroids, data, labels, m, n, k);

    printConfusionMatrix(labels, results);
}*/

void run_iris_data(int itr_){

    int m = 150;
    int n = 4;
    int k = 3;
    int itr = itr_;

    double *centroids = (double*) malloc(k*n*sizeof(double));
    kmeans(data, m, n, k, centroids, itr);

    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }


}


int main(){

 //  run_small_kmeans_test();
 //  run_uni_data_test();
    for(int i = 0; i < 20; i++){
        printf("\nIteration %d------\n", i);
        run_iris_data(i);
    }
   return 0;
} 
