#include "kmeansHelper.cu"
#include "hip/hip_runtime.h"
#include <stdio.h>


void kmeans(double* data, int m, int n, int k, double* centroids, int iterations){
    
    
    double *data_d;
    double *centroids_d;
    int *counts;
    int *labels;
    double *distances;
    
    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&counts, k*sizeof(int));
    hipMalloc((void**)&labels, m*sizeof(int));
    hipMalloc((void**)&distances, k*sizeof(double));

    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);
    
    init_zero<<<k, n>>>(centroids);

    // Set number of iterations
    for(int step__ = 0; step__ < iterations; step__++){ 

        // Assignment Step
        for(int i = 0; i < m; i++){

            subtractPointFromMeans<<<k, n>>>(data, centroids, m, n, k, i);

            getDistances<<<k, 1>>>(centroids, distances, k, n);

            assignClass(distances, labels, k, i);

            addPointToMeans<<<k, n>>>(data, centroids, m, n, k, i);

        }            
        
        // Update Means Step
        init_zero<<<k, 1>>>(counts);

        findNewCentroids<<<k, n>>>(points, centroids, labels, m, n, k, counts);

        divide_by_count<<<k, n>>>(centroids, counts, n, k);

    }

}
