#include "kmeansHelper.cu"
#include "hip/hip_runtime.h"
#include <stdio.h>


void kmeans(double* data, int m, int n, int k, double* centroids, int iterations){
    
    
    double *data_d;
    double *centroids_d;
    int *counts;
    int *labels;
    double *distances;
    
    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&counts, k*sizeof(int));
    hipMalloc((void**)&labels, m*sizeof(int));
    hipMalloc((void**)&distances, k*sizeof(double));

    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);

    init_labels<<<m, 1>>>(labels, k);
    
    // Update Means Step
    init_zeros<<<k, 1>>>(counts);
    init_zeros<<<k, n>>>(centroids_d);

    findNewCentroids<<<m, n>>>(data_d, centroids_d, labels, m, n, k, counts);
    
    divide_by_count<<<k, n>>>(centroids_d, counts, n, k);

    hipMemcpy(centroids, centroids_d, k*n*sizeof(double), hipMemcpyDeviceToHost);

    // Set number of iterations
    for(int step__ = 0; step__ < iterations; step__++){ 

        // Assignment Step
        for(int point = 0; point < m; point++){

            subtractPointFromMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);

            getDistances<<<k, 1>>>(centroids_d, distances, k, n);

            assignClass<<<1, 1>>>(distances, labels, k, point);

            addPointToMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);

        }            
        
        // Update Means Step
        init_zeros<<<k, 1>>>(counts);
        init_zeros<<<k, n>>>(centroids_d);


        findNewCentroids<<<m, n>>>(data_d, centroids_d, labels, m, n, k, counts);

        divide_by_count<<<k, n>>>(centroids_d, counts, n, k);

    }
    hipMemcpy(centroids, centroids_d, k*n*sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }

}

void run_small_kmeans_test(){


    int m = 12;
    int n = 2;
    int k = 3;
    int iterations = 5;

    double data[m*n] = {0, 1, 1, 0, 1, 1,0,0, 5, 6, 6, 7,7,5 ,5, 5, 0, 8, 1, 9, 0, 9,1,8};
    double* centroids = (double*) malloc(k*n*sizeof(double));

    kmeans(data, m, n, k, centroids, iterations);

}

int main(){
    
   run_small_kmeans_test(); 
   return 0;
}
