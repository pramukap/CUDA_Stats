#include "hip/hip_runtime.h"
#include "kmeansHelper.cu"
//#include "University_Data.h"
#include "Iris_Data.h"
//#include "large_cluster.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" void kmeans(double* data, int m, int n, int k, double* centroids, int iterations);

extern "C" {
// Assume data is filled
// Assume centroids is allocated
void kmeans(double* data, int m, int n, int k, double* centroids, int iterations){

    printf("RUNNING KMEANS: (%dx%d) for %d clusters using %d itr\n", m, n, k, iterations);

    double *data_d;
    double *centroids_d;
    int *counts;
    int *labels;
    double *distances;

    double *new_mean;

    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&new_mean, k*n*sizeof(double));
    hipMalloc((void**)&counts, k*sizeof(int));
    hipMalloc((void**)&labels, m*sizeof(int));
    /* old distance
    hipMalloc((void**)&distances, k*sizeof(double));
    */
    hipMalloc((void**)&distances, m*sizeof(double));
    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);

/*
    for(int i = 0; i < m*n; i++){
        printf("%f\t", data[i]);
    }
*/

    // Initalize centroids using random partition of data into k groups
    init_labels<<<m, 1>>>(labels, k);
    init_zeros<<<k, 1>>>(counts);
    init_zeros<<<k, n>>>(centroids_d);

    findNewCentroids<<<m, n>>>(data_d, centroids_d, labels, m, n, k, counts);
    divide_by_count<<<k, n>>>(centroids_d, counts, n, k);


    // Set number of iterations
    for(int step__ = 0; step__ < iterations; step__++){
/*
        hipMemcpy(centroids, centroids_d, k*n*sizeof(double), hipMemcpyDeviceToHost);
        for(int i = 0; i < k*n; i++){
            printf("%f\t",centroids[i]);
        }
        printf("\n\n");
*/
        // Assignment Step
        init_zeros<<<m, 1>>>(distances);
        assignClasses<<<m, 1>>>(data_d, centroids_d, m, n, k, labels, distances);

/*
        // OLD ASSIGNMENT
        for(int point = 0; point < m; point++){

            subtractPointFromMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);
            getDistances<<<k, 1>>>(centroids_d, distances, k, n);
            assignClass<<<1, 1>>>(distances, labels, k, point);
            addPointToMeans<<<k, n>>>(data_d, centroids_d, m, n, k, point);

        }
*/        // Update Means Step
        init_zeros<<<k, 1>>>(counts);
        init_zeros<<<k, n>>>(new_mean);
        findNewCentroids<<<m, n>>>(data_d, new_mean, labels, m, n, k, counts);
        divide_by_count<<<k, n>>>(new_mean, counts, n, k);

        copyCentroidToOld<<<k, n>>>(new_mean, centroids_d);

    }
    hipMemcpy(centroids, centroids_d, k*n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(centroids_d);
    hipFree(counts);
    hipFree(labels);
    hipFree(distances);

}

// Assume centroids is filled
// Assume labels is allocated
// Assume data is filled
void kmeans_classify(double * centroids, double * data, int *labels_h, int m, int n, int k){

    double *data_d;
    double *centroids_d;
    int *labels;
    double *distances;
/*
    printf("RUNNING CLASSIFY m%d\tn%d\tk%d\t\n", m, n, k);

    for(int i = 0; i < k*n; i++){
        printf("%f\t",centroids[i]);
    }
    for(int i = 0; i < m*n; i++){
        printf("%f\t",data[i]);
    }
    for(int i = 0; i < m; i++){
        printf("%d\t",labels_h[i]);
    }
*/
    hipMalloc((void**)&data_d, m*n*sizeof(double));
    hipMalloc((void**)&centroids_d, k*n*sizeof(double));
    hipMalloc((void**)&labels, m*sizeof(int));
    hipMalloc((void**)&distances, k*sizeof(double));

    hipMemcpy(data_d, data, m*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(centroids_d, centroids, k*n*sizeof(double), hipMemcpyHostToDevice);

    printf("RUNNING CLASSIFY\n");

    // Assignment Step
    init_zeros<<<m, 1>>>(distances);
    assignClasses<<<m, 1>>>(data_d, centroids_d, m, n, k, labels, distances);

    hipMemcpy(labels_h, labels, m*sizeof(int), hipMemcpyDeviceToHost);

/*    for(int i = 0; i < m; i++){
        printf("%d\t",labels_h[i]);
    }
*/
}
}
void run_small_kmeans_test(){

    int m = 12;
    int n = 2;
    int k = 3;
    int iterations = 100;

    double data[m*n] = {0, 1, 1, 0, 1, 1,0,0, 5, 6, 6, 7,7,5 ,5, 5, 0, 8, 1, 9, 0, 9,1,8};
    double* centroids = (double*) malloc(k*n*sizeof(double));

    kmeans(data, m, n, k, centroids, iterations);

    for(int i = 0; i < k*n; i++){
        printf("%f\n", centroids[i]);
    }

}

void printConfusionMatrix(int *actual, int*expect){

    int tp = 0;
    int fp = 0;
    int fn = 0;
    int tn = 0;

    for(int i = 0; i < 777; i++){
        if(actual[i] == expect[i]){
            if(actual[i] == 1){
                tp++;
            } else {
                tn++;
            }
        } else {
            if (actual[i] == 1){
                fp++;
            }else{
                fn++;
            }
        }
    }
    printf("\n");
    printf("TP: %d\nFP: %d\nFN: %d\nTN: %d\n", tp, fp, fn, tn);


}
/*
void run_uni_data_test(){

    int m = 777;
    int n = 17;
    int k = 2;
    int iterations = 10;

    double* centroids = (double*) malloc(k*n*sizeof(double));
    kmeans(data, m, n, k, centroids, iterations);
    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }

    int *labels = (int *) malloc(sizeof(int) * m);
    kmeans_classify(centroids, data, labels, m, n, k);

    printConfusionMatrix(labels, results);
}*/

void run_iris_data(int itr_){

    int m = 150;
    int n = 4;
    int k = 3;
    int itr = itr_;

    double *centroids = (double*) malloc(k*n*sizeof(double));
    kmeans(data, m, n, k, centroids, itr);

    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }


}

void run_iris_data_output(int itr_){

    int m = 150;
    int n = 4;
    int k = 3;
    int itr = itr_;

    double *centroids = (double*) malloc(k*n*sizeof(double));
    kmeans(data, m, n, k, centroids, itr);

    for(int i = 0; i < k; i++){
        if(i==2){
        for(int j = 0; j < n; j++){
            if(j != n -1){
            printf("%f,", centroids[i*n + j]);
            } else {
                printf("%f", centroids[i*n + j]);
            }
        }}
    }
    int *labels = (int *) malloc(sizeof(int) * m);
    kmeans_classify(centroids, data, labels, m, n, k);
    for(int i = 0; i < m; i++){
        if (i == m-1){
            printf("%d", labels[i]);
        } else{
        printf("%d,", labels[i]);
        }}


}

void run_large_dataset(int m_){

    int m = m_;
    int n = 2;
    int k = 15;
    int itr = 500;

    clock_t start, end;
    double cpu_time;

    double *centroids = (double*) malloc(k*n*sizeof(double));

    start = clock();
    kmeans(data, m, n, k, centroids, itr);
    end = clock();

    cpu_time = ((double)(end-start))/CLOCKS_PER_SEC;

    printf("Size: %d ran in %f\n", m, cpu_time);
    for(int i = 0; i < k; i++){
        printf("\nKmean%d:\t", i);
        for(int j = 0; j < n; j++){
            printf("%f\t", centroids[i*n + j]);
        }
    }
    
}

int main(){
    run_iris_data_output(20);
    return 0;
}
/*
int main(){
   
    
    run_large_dataset(10);
    run_large_dataset(100);
    run_large_dataset(500);
    run_large_dataset(1000);
    run_large_dataset(2500);
    run_large_dataset(5000);
 //  run_small_kmeans_test();
 //  run_uni_data_test();
//    for(int i = 0; i < 12; i++){
 //       printf("\n");
   //     run_iris_data_output(i);
   // }
   return 0;
}*/ 
