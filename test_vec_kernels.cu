#include "hip/hip_runtime.h"
#include    "hip/hip_runtime.h"
#include    "vec_kernels.cuh"

#include    <cassert>
#include    <cmath>
#include    <iostream>

int     main(void) 
{
    std::cout << "Test: vec_dot_product" << std::endl;
    
    double *a, *b, *out;
    hipMallocManaged(&a, 1024 * sizeof(double));
    hipMallocManaged(&b, 1024 * sizeof(double));
    hipMallocManaged(&out, sizeof(double));
    for (size_t i = 0; i < 1024; i++) {
        a[i] = 1.0;
        b[i] = 1.0;
    }

    vec_dot_product<<<1, 1024, sizeof(double) * 1024>>>(a, b, out, 1, 1024);
    hipDeviceSynchronize();
    assert(abs(*out - 1024.0) <= 1.0);

    hipFree(a);
    hipFree(b);
    hipFree(out);
    std::cout << "Passed: vec_dot_product" << std::endl;
    std::cout << "Test: vec_scalar_mul" << std::endl;

    hipMallocManaged(&a, 1024 * sizeof(double));
    hipMallocManaged(&b, 1024 * sizeof(double));
    
    for (size_t i = 0; i < 1024; i++) {
        a[i] = b[i] = 1.0;
    }

    vec_scalar_mul<<<1, 1024>>>(a, b, 1024, 1, 1024);
    hipDeviceSynchronize();
    for (size_t i = 0; i < 1024; i++) {
        assert(abs(b[i] - 1024.0) <= 1.0);
    }
    std::cout << "Passed: vec_scalar_mul" << std::endl;

    hipFree(a);
    hipFree(b);
    return 0;
}
