#include "hip/hip_runtime.h"
#include    "hip/hip_runtime.h"
#include    "vec_kernels.cuh"
#include	"matrixFunctions.cuh"
#include    "math.h" 

#include    <cstddef>
#include	<cstdlib>
#include    <iostream>

#define     BLOCKSIZE       1024
#define     GRIDSIZE(d)     (((d) + ((BLOCKSIZE) - 1)) / (BLOCKSIZE))

__global__
void    mat_transpose(double *X, double *Xt, size_t m, size_t n) 
{
    size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= m*n)
        return;
    
    size_t row = gid / n;
    size_t col = gid % n;
    Xt[col * m + row] = X[row * n + col];
}

extern "C"
{

void    fit(double *X, double *y, double *theta, double lr, size_t m, size_t n, size_t n_iter) 
{
    double *Xt, *Xd, *yd, *thetad;
    
    hipMalloc(&Xt, sizeof(double) * n * m);
    
    hipMalloc(&Xd, sizeof(double) * n * m);
    hipMemcpy(Xd, X, sizeof(double) * n * m, hipMemcpyHostToDevice);
    
    hipMalloc(&yd, sizeof(double) * m);
    hipMemcpy(yd, y, sizeof(double) * m, hipMemcpyHostToDevice);
    
    hipMalloc(&thetad, sizeof(double) * n);
    hipMemcpy(thetad, theta, sizeof(double) * n, hipMemcpyHostToDevice);
    
    mat_transpose<<<GRIDSIZE(m*n), BLOCKSIZE>>>(X, Xt, m, n);
    hipDeviceSynchronize();

    for (size_t i = 0; i < n_iter; i++) {
        double *z, *h, *g;
        hipMallocManaged(&z, sizeof(double) * m);
        hipMallocManaged(&h, sizeof(double) * m);
        hipMallocManaged(&g, sizeof(double) * n);

        // dot(X, theta)
        vec_dot_mat<<<GRIDSIZE(m), BLOCKSIZE>>>(X, theta, z, m, n);
        hipDeviceSynchronize();

        // h = sigm(z)
        vec_sigmoid<<<GRIDSIZE(m), BLOCKSIZE>>>(z, h, 1, m);
        hipDeviceSynchronize();

        // h = -h
        vec_scalar_mul<<<GRIDSIZE(m), BLOCKSIZE>>>(h, h, -1.0, 1, m);
        hipDeviceSynchronize();

        // h = y - h
        vec_add<<<GRIDSIZE(m), BLOCKSIZE>>>(h, y, h, 1, m);
        hipDeviceSynchronize();

        // h = -(y - h) = h - y
        vec_scalar_mul<<<GRIDSIZE(m), BLOCKSIZE>>>(h, h, -1.0, 1, m); 

        // g = dot(Xt, h)
        vec_dot_mat<<<GRIDSIZE(n), BLOCKSIZE>>>(Xt, h, g, n, m);
        hipDeviceSynchronize();

        // g = -(g*lr) / m
        vec_scalar_mul<<<GRIDSIZE(n), BLOCKSIZE>>>(g, g, -(lr / m), 1, n);
        hipDeviceSynchronize();

        // theta = theta + (-g) = theta - g
        vec_add<<<GRIDSIZE(n), BLOCKSIZE>>>(theta, g, theta, 1, n);
        hipDeviceSynchronize();

        hipFree(z);
        hipFree(h);
        hipFree(g);
    }

    hipFree(Xd);
    hipFree(Xt);
    hipFree(yd);
    hipMemcpy(theta, thetad, sizeof(double) * n, hipMemcpyDeviceToHost);
    hipFree(thetad);
}

void    predict_proba(double *X, double *theta, double *y, size_t m, size_t n)
{
    double *yd;
    double *Xd;
    double *thetad;
    hipMalloc(&yd, sizeof(double) * m);
    hipMalloc(&Xd, sizeof(double) * m * n);
    hipMalloc(&thetad, sizeof(double) * n);
    hipMemcpy((void*) Xd, (void*) X, sizeof(double) * m * n, hipMemcpyHostToDevice);
    hipMemcpy((void*) thetad, (void*) theta, sizeof(double) * n, hipMemcpyHostToDevice);

    MatrixMul<<<m, n>>>(Xd, thetad, yd, n, m, 1, n);
    hipDeviceSynchronize();

    hipMemcpy((void*) y, yd, sizeof(double) * m, hipMemcpyDeviceToHost);
    hipFree(Xd);
    hipFree(thetad);
    hipFree(yd);
    return y;
}

}

int	main(void)
{
	double *X; // = (double*) malloc(sizeof(double) * 1024 * 1024);
	double *y; // = (double*) malloc(sizeof(double) * 1024);
	double *theta; // = (double*) malloc(sizeof(double) * 1024);

	hipMallocManaged(&X, sizeof(double) * 1024 * 1024);
	hipMallocManaged(&y, sizeof(double) * 1024);
	hipMallocManaged(&theta, sizeof(double) * 1024);
	for (int i = 0; i < 1024; i++) {
		y[i] = i % 2;
		theta[i] = 0;
		for (int j = 0; j < 1024; j++) {
			X[i * 1024 + j] = j;
		}
	}

	fit(X, y, theta, 0.01, 1024, 1024, 1);
}


