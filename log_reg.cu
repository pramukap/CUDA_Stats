#include    "hip/hip_runtime.h"
#include    "matrixFunctions.cuh"
#include    "vec_kernels.cuh"
#include    "math.h" 

#include    <cstddef>
#include    <iostream>

#define     BLOCKSIZE       1024
#define     GRIDSIZE(d)     (((d) + ((BLOCKSIZE) - 1)) / (BLOCKSIZE))

extern "C"
{

void    fit(double *X, double *y, double *theta, double lr, size_t n, size_t m, size_t n_iter) 
{
    double *Xt, *Xd, *yd, *thetad;
    
    hipMalloc(&Xt, sizeof(double) * n * m);
    
    hipMalloc(&Xd, sizeof(double) * n * m);
    hipMemcpy(Xd, X, sizeof(double) * n * m, hipMemcpyHostToDevice);
    
    hipMalloc(&yd, sizeof(double) * m);
    hipMemcpy(yd, y, sizeof(double) * m, hipMemcpyHostToDevice);
    
    hipMalloc(&thetad, sizeof(double) * n);
    hipMemcpy(thetad, theta, sizeof(double) * n, hipMemcpyHostToDevice);
    MatrixTranspose<<<n, m>>>(Xd, Xt, m, n);
    hipDeviceSynchronize();

    for (size_t i = 0; i < n_iter; i++) {
        double *z, *h, *g;
        hipMalloc(&z, sizeof(double) * m);
        hipMalloc(&h, sizeof(double) * m);
        hipMalloc(&g, sizeof(double) * n);

        MatrixMul<<<n, n>>>(X, theta, z, n, m, 1, n);
        hipDeviceSynchronize();

        vec_sigmoid<<<GRIDSIZE(n), BLOCKSIZE>>>(z, h, 1, m);
        hipDeviceSynchronize();

        vec_scalar_mul<<<GRIDSIZE(n), BLOCKSIZE>>>(h, h, -1.0, 1, m);
        hipDeviceSynchronize();

        vec_add<<<GRIDSIZE(n), BLOCKSIZE>>>(h, y, h, 1, m);
        hipDeviceSynchronize();

        MatrixMul<<<m, m>>>(Xt, h, g, m, n, 1, m);
        hipDeviceSynchronize();

        vec_scalar_mul<<<GRIDSIZE(n), BLOCKSIZE>>>(g, g, -lr / m, 1, m);
        hipDeviceSynchronize();

        vec_add<<<GRIDSIZE(n), BLOCKSIZE>>>(theta, g, theta, 1, m);
        hipDeviceSynchronize();

        hipFree(z);
        hipFree(h);
        hipFree(g);
    }

    hipFree(Xd);
    hipFree(Xt);
    hipFree(yd);
    hipMemcpy(theta, thetad, sizeof(double) * n, hipMemcpyDeviceToHost);
    hipFree(thetad);
}

double*    predict_proba(double *X, double *theta, size_t m, size_t n)
{
    double *y = (double*) malloc(sizeof(double) * m);
    double *yd;
    double *Xd;
    double *thetad;
    hipMalloc(&yd, sizeof(double) * m);
    hipMalloc(&Xd, sizeof(double) * m * n);
    hipMalloc(&thetad, sizeof(double) * n);
    hipMemcpy((void*) Xd, (void*) X, sizeof(double) * m * n, hipMemcpyHostToDevice);
    hipMemcpy((void*) thetad, (void*) theta, sizeof(double) * n, hipMemcpyHostToDevice);

    MatrixMul<<<GRIDSIZE(m * n), BLOCKSIZE>>>(Xd, thetad, yd, n, m, 1, n);
    hipDeviceSynchronize();

    hipMemcpy((void*) y, yd, sizeof(double) * m, hipMemcpyDeviceToHost);
    hipFree(Xd);
    hipFree(thetad);
    hipFree(yd);
    return y;
}

}


