#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// define matrix size
#define AX 3
#define AY 3
#define BX 4
#define BY 4


// possible matrix struct, didnt use here
struct Matrix {

	int col;
	int row;
	double * data;
};

// inverts a matrix A by turning first N columns of A|I into RREF
// # threads = 2N

//TODO: write rule for swapping
//TODO: write function to concatenate identity matrix to the end of A
//              effectively     A:-> A|I

// each thread corresponds to a particular column

// perform division on row to turn leading nonzero into a 1
// perform elimination on all other rows to make pivot column 0s
__global__ void MatrixInverse(double *A, int Ax, int Ay) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double mult;
	double to_mult;
	double old_val;
	int current_pivot_col = 0;
	int i = 0;
	for (i = 0; i < Ax; i++) {
		// SWAP CODE
		if (i == col && A[i*Ay + col] == 0) {
			for (int k = i; k < Ax; k++) {
				if (A[k*Ay + col] != 0) {
					for (int x = 0; x < Ay; x++) {
						int tmp = A[i*Ay + x];
						A[i*Ay + x] = A[k*Ay + x];
						A[k*Ay + x] = tmp;
					}
					break;
				}

			}
		}

		// divide element by pivot
		__syncthreads();
		A[i*Ay + col] = A[i*Ay + col] / A[i*Ay + i];
		__syncthreads();

		for (int j = 0; j < Ax; j++) {
			mult = A[j*Ay + i];
			to_mult = A[i*Ay + col];
			old_val = A[j*Ay + col];
			//printf("mult = %f index = %d, to_mult = %f index = %d, old_val = %f index = %d, thread = %d, j = %d, i = %d, col = %d, Ay = %d\n", mult, (j*Ay + i), to_mult, (i*Ay + col), old_val, (j*Ay + col), col, j, i, col, Ay);
			if ((j != i) && (A[j*Ay + i] != 0)) {
				A[j*Ay + col] = old_val - mult * to_mult;
			}
		}

		__syncthreads();
	}
}

// Function that appends an identity matrix to the right of the current matrix
// keeping new matrix in row major form
// constant time in parallel
// assume that dst has 2*N*N = 2*len(src) allocated
__global__ void MatrixAppendIdentity(double* src, double* dst, int num_row, int num_col) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i % (2 * num_col) < num_col) {
		dst[i] = src[(num_row*(i / (2 * num_row))) + (i % (2 * num_row))];
	}
	else if ((i % (2 * num_row) - num_row == i / (2 * num_row))) {
		dst[i] = 1;
	}
	else {
		dst[i] = 0;
	}

}

__global__ void ExtractInverse(double *src, double* dst, int num_row, int num_col) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i % (2 * num_col) >= num_col) {
		dst[(num_row*(i / (2 * num_row))) + (i % (2 * num_row) - num_row)] = src[i];
	}


}

// adds arrays A and B and stores the result in C 
// assume all arrays have the same dimensions
__device__ void MatrixAdd(double * A, double * B, double * C) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	C[x] = A[x] + B[x];
}

// performs scalar multiplication on matrix A and scalar X
// stores result in B
__device__ void MatrixSMul(double * A, double * B, double scalar) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	B[x] = A[x] * scalar;
}

// transpose function, A is input, B is output, Ax and Ay are the dimensions of A
__device__ void MatrixTranspose(double * A, double * B, int Ax, int Ay) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int new_row, new_loc;
	if (x == 0) {
		new_loc = 0;
	}
	else {
		new_row = (x % Ax) * Ay;
		new_loc = new_row + (x / Ax);
	}

	B[new_loc] = A[x];
}

// multiplies the matrices A and B and stores them into C
// Ax, Ay, Bx, By are the dimensions
// use a thread for each element of the final C array.
__device__ void MatrixMul(double * A, double * B, double * C, int Ax, int Ay, int Bx, int By) {
	if (Ax == By) {
		// total array position
		int x = blockIdx.x * blockDim.x + threadIdx.x;

		int count;
		int Aindex, Bindex;
		double prod;
		for (count = 0; count < Ax; count++) {
			// row of C matrix
			Aindex = (x / Bx) * Ax + count;
			// column of C matrix
			Bindex = (x % Bx) + By * count;
			prod = A[Aindex] * B[Bindex];
			C[x] += prod;
		}
	}
}

// kernel that calls the function
__global__ void MatrixKernel(double * A, double * B, double * C, int Ax, int Ay, int Bx, int By) {
	//MatrixAppendIdentity(A, C, 4, 4);
	//MatrixMul(A, B, C, Ax, Ay, Bx, By);
	//MatrixAdd(A, B, C);
	//MatrixSMul(A, C, 10);
}

int main()
{
	int Asize = AX * AY * sizeof(double);
	int Bsize = BX * BY * sizeof(double);
	int Csize = AX * BY * sizeof(double);
	int AarrSize = AX * AY;
	int BarrSize = BX * BY;
	int CarrSize = BX * AY;
	double * MatA = (double *)malloc(Asize);
	double * MatB = (double *)malloc(Bsize);
	double * MatC = (double *)malloc(Csize);
	double * MatD = (double *)malloc(2 * Asize);
	double * MatA_d;
	double * MatB_d;
	double * MatC_d;
	double * MatD_d;

	hipMalloc((void **)&MatA_d, Asize);
	hipMalloc((void **)&MatB_d, Bsize);
	hipMalloc((void **)&MatC_d, Csize);
	hipMalloc((void **)&MatD_d, 2 * Asize);

	// set up array
	double Mat[9] = { 1, 2, 3, 0, 1, 4, 5, 6, 0 };
	memcpy(MatA, Mat, 9 * sizeof(double));

	// print initial array
	int x;
	for (x = 0; x < AarrSize; x++) {
		//MatA[x] = x;
		printf("%d ", (int)MatA[x]);
		if (x != 0) {
			if ((x % AX) == (AX - 1)) {
				printf("\n");
			}
		}
	}
	printf("\n");
	hipMemcpy(MatA_d, MatA, Asize, hipMemcpyHostToDevice);

	//for (x = 0; x < BarrSize; x++) {
	//MatB[x] = x;
	//printf("%d ", (int)MatB[x]);
	//if (x != 0) {
	//if ((x % BX) == (BX - 1)) {
	//printf("\n");
	//}
	//}
	//}
	//printf("\n");
	//hipMemcpy(MatB_d, MatB, Bsize, hipMemcpyHostToDevice);

	//MatrixKernel << <AX, 2 * AY >> > (MatA_d, MatB_d, MatD_d, AX, AY, BX, BY);

	// append identity and print
	MatrixAppendIdentity << <AX, 2 * AY >> > (MatA_d, MatD_d, AX, AY);
	hipMemcpy(MatD, MatD_d, 2 * Asize, hipMemcpyDeviceToHost);
	for (x = 0; x < (2 * AarrSize); x++) {
		printf("%d ", (int)MatD[x]);
		if (x != 0) {
			if ((x % (2 * AX)) == ((2 * AX) - 1)) {
				printf("\n");
			}
		}
	}
	printf("\n");

	// invert and print
	MatrixInverse << <1, 2 * AX >> > (MatD_d, AY, 2 * AX);
	hipMemcpy(MatD, MatD_d, 2 * Asize, hipMemcpyDeviceToHost);
	for (x = 0; x < (2 * AarrSize); x++) {
		printf("%f ", MatD[x]);
		if (x != 0) {
			if ((x % (2 * AX)) == ((2 * AX) - 1)) {
				printf("\n");
			}
		}
	}
	printf("\n");

	// extract inverse and print
	ExtractInverse << <AX, 2 * AY >> > (MatD_d, MatA_d, AX, AY);
	hipMemcpy(MatA, MatA_d, Asize, hipMemcpyDeviceToHost);
	for (x = 0; x < (AarrSize); x++) {
		printf("%d ", (int)MatA[x]);
		if (x != 0) {
			if ((x % (AX)) == ((AX)-1)) {
				printf("\n");
			}
		}
	}

	return 0;
}
