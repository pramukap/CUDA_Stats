#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// adds arrays A and B and stores the result in C 
// assume all arrays have the same dimensions
__global__ void MatrixAdd(double * A, double * B, double * C) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	C[x] = A[x] + B[x];
}

// performs scalar multiplication on matrix A and scalar X
// stores result in B
__global__ void MatrixSMul(double * A, double * B, double X) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	B[x] = A[x] * X;
}

// transpose function, A is input, B is output, Ax and Ay are the dimensions of A
__global__ void MatrixTranspose(double * A, double * B, int Ax, int Ay) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int new_row, new_loc;
	if (x == 0) {
		new_loc = 0;
	}
	else {
		new_row = (x % Ax) * Ay;
		new_loc = new_row + (x / Ax);
	}

	B[new_loc] = A[x];
}

// multiplies the matrices A and B and stores them into C
// Ax, Ay, Bx, By are the dimensions
// use a thread for each element of the final C array.
__global__ void MatrixMul(double * A, double * B, double * C, int Ax, int Ay, int Bx, int By) {
	if (Ax == By) {

		// total array position
		int x = blockIdx.x * blockDim.x + threadIdx.x;

		// reset C array
		C[x] = 0;
		__syncthreads();

		int count;
		int Aindex, Bindex;
		double prod;
		for (count = 0; count < Ax; count++) {
			// row of C matrix
			Aindex = (x / Bx) * Ax + count;
			// column of C matrix
			Bindex = (x % Bx) + Bx * count;
			prod = A[Aindex] * B[Bindex];
			C[x] += prod;
		}
	}
}

// inverts a matrix A by turning first N columns of A|I into RREF
// # threads = 2N

//TODO: write rule for swapping
//TODO: write function to concatenate identity matrix to the end of A
//              effectively     A:-> A|I

// each thread corresponds to a particular column

// perform division on row to turn leading nonzero into a 1
// perform elimination on all other rows to make pivot column 0s
// call so Ax = AY and Ay = 2 * AX
__global__ void MatrixInverse(double *A, int Ax, int Ay) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double mult;
	double to_mult;
	double old_val;
	int current_pivot_col = 0;
	int i = 0;
	for (i = 0; i < Ax; i++) {
		// SWAP CODE
		if (i == col && A[i*Ay + col] == 0) {
			for (int k = i; k < Ax; k++) {
				if (A[k*Ay + col] != 0) {
					for (int x = 0; x < Ay; x++) {
						int tmp = A[i*Ay + x];
						A[i*Ay + x] = A[k*Ay + x];
						A[k*Ay + x] = tmp;
					}
					break;
				}

			}
		}

		// divide element by pivot
		__syncthreads();
		A[i*Ay + col] = A[i*Ay + col] / A[i*Ay + i];
		__syncthreads();

		for (int j = 0; j < Ax; j++) {
			mult = A[j*Ay + i];
			to_mult = A[i*Ay + col];
			old_val = A[j*Ay + col];
			if ((j != i) && (A[j*Ay + i] != 0)) {
				A[j*Ay + col] = old_val - mult * to_mult;
			}
		}

		__syncthreads();
	}
}

// Function that appends an identity matrix to the right of the current matrix
// keeping new matrix in row major form
// constant time in parallel
// assume that dst has 2*N*N = 2*len(src) allocated
__global__ void MatrixAppendIdentity(double* src, double* dst, int num_row, int num_col) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i % (2 * num_col) < num_col) {
		dst[i] = src[(num_row*(i / (2 * num_row))) + (i % (2 * num_row))];
	}
	else if ((i % (2 * num_row) - num_row == i / (2 * num_row))) {
		dst[i] = 1;
	}
	else {
		dst[i] = 0;
	}

}



__global__ void ExtractInverse(double *src, double* dst, int num_row, int num_col){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i % (2*num_col) >= num_col){
		dst[(num_row*(i / (2 * num_row))) + (i % (2 * num_row) - num_row)] = src[i];
    }


}

// takes an array of doubles and its dimensions as input
// sets the array to (((A^t)(A))^-1)(A^t)B
// where A is a matrix with Ay elements each having Ax features
// and B is a vector containing Ay elements
// C is a vector with Ax elements
void get_beta(double * A, double * B, double * C, int Ax, int Ay) {
	int x;
	double * MatA = (double *)malloc(Ax * Ay * sizeof(double));
	double * MatB = (double *)malloc(Ax * Ay * sizeof(double));
	double * MatC = (double *)malloc(Ax * Ax * sizeof(double));
	double * MatD = (double *)malloc(2 * Ax * Ax * sizeof(double));
	double * MatA_d;
	double * MatB_d;
	double * MatC_d;
	double * MatD_d;
	double * MatE_d;
	double * Beta_d;
	hipMalloc((void **)&MatA_d, Ax * Ay * sizeof(double));
	hipMalloc((void **)&MatB_d, Ax * Ay * sizeof(double));
	hipMalloc((void **)&MatC_d, Ax * Ax * sizeof(double));
	hipMalloc((void **)&MatD_d, 2 * Ax * Ax * sizeof(double));
	hipMalloc((void **)&MatE_d, Ay * sizeof(double));
	hipMalloc((void **)&Beta_d, Ax * sizeof(double));
	hipMemcpy(MatA_d, A, Ax * Ay * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(MatE_d, B, Ay * sizeof(double), hipMemcpyHostToDevice);

	// B = Transpose(A)
	MatrixTranspose << < Ay, Ax >> > (MatA_d, MatB_d, Ax, Ay);

	// C = BA
	MatrixMul << <Ax, Ax >> > (MatB_d, MatA_d, MatC_d, Ay, Ax, Ax, Ay);

	// Invert C
	MatrixAppendIdentity << <Ax, 2 * Ax >> > (MatC_d, MatD_d, Ax, Ax);
	MatrixInverse << <1, 2 * Ax >> > (MatD_d, Ax, 2 * Ax);
	ExtractInverse << <Ax, 2 * Ax >> > (MatD_d, MatC_d, Ax, Ax);

	// A = CB
	MatrixMul << <Ax, Ay >> > (MatC_d, MatB_d, MatA_d, Ax, Ax, Ay, Ax);

	// Beta = AE
	MatrixMul << <1, Ax >> > (MatA_d, MatE_d, Beta_d, Ay, Ax, 1, Ay);

	// return Beta
	hipMemcpy(C, Beta_d, Ax * sizeof(double), hipMemcpyDeviceToHost);

	// free resources
	free(MatA);
	free(MatB);
	free(MatC);
	free(MatD);
	hipFree(MatA_d);
	hipFree(MatB_d);
	hipFree(MatC_d);
	hipFree(MatD_d);
	hipFree(MatE_d);
	hipFree(Beta_d);
}

// Performs matrix multiplication on A and B
// A a matrix of known values with Ay rows and Ax columns
// B is the beta vector with Ax values
// C is the output vector with Ay values
void linreg(double * A, double * B, double * C, int Ax, int Ay) {
	double * MatA = (double *)malloc(Ax * Ay * sizeof(double));
	double * MatB = (double *)malloc(Ax * sizeof(double));
	double * MatC = (double *)malloc(Ay * sizeof(double));
	double * MatA_d;
	double * MatB_d;
	double * MatC_d;
	hipMalloc((void **)&MatA_d, Ax * Ay * sizeof(double));
	hipMalloc((void **)&MatB_d, Ax * Ay * sizeof(double));
	hipMalloc((void **)&MatC_d, Ax * Ax * sizeof(double));
	hipMemcpy(MatA_d, A, Ax * Ay * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(MatB_d, B, Ax * sizeof(double), hipMemcpyHostToDevice);

	// C = AB
	MatrixMul << <1, Ay >> > (MatA_d, MatB_d, MatC_d, Ax, Ay, 1, Ax);

	// return C
	hipMemcpy(C, MatC_d, Ay * sizeof(double), hipMemcpyDeviceToHost);

	// free resources
	free(MatA);
	free(MatB);
	free(MatC);
	hipFree(MatA_d);
	hipFree(MatB_d);
	hipFree(MatC_d);
}
