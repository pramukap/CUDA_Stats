#include "hip/hip_runtime.h"

#include <stdio.h>

// adds arrays A and B and stores the result in C 
// assume all arrays have the same dimensions
__device__ void MatrixAdd(double * A, double * B, double * C) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	C[x] = A[x] + B[x];
}

// performs scalar multiplication on matrix A and scalar X
// stores result in B
__device__ void MatrixSMul(double * A, double * B, double X) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	B[x] = A[x] * X;
}

// transpose function, A is input, B is output, Ax and Ay are the dimensions of A
__device__ void MatrixTranspose(double * A, double * B, int Ax, int Ay) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int new_row, new_loc;
	if (x == 0) {
		new_loc = 0;
	}
	else {
		new_row = (x % Ax) * Ay;
		new_loc = new_row + (x / Ax);
	}

	B[new_loc] = A[x];
}

// multiplies the matrices A and B and stores them into C
// Ax, Ay, Bx, By are the dimensions
// use a thread for each element of the final C array.
__global__ void MatrixMul(double * A, double * B, double * C, int Ax, int Ay, int Bx, int By) {
	if (Ax == By) {
		// total array position
		int x = blockIdx.x * blockDim.x + threadIdx.x;

		int count;
		int Aindex, Bindex;
		double prod;
		for (count = 0; count < Ax; count++) {
			// row of C matrix
			Aindex = (x / Bx) * Ax + count;
			// column of C matrix
			Bindex = (x % Bx) + By * count;
			prod = A[Aindex] * B[Bindex];
			C[x] += prod;
		}
	}
}

// inverts a matrix A by turning first N columns of A|I into RREF
// # threads = 2N

//TODO: write rule for swapping
//TODO: write function to concatenate identity matrix to the end of A
//              effectively     A:-> A|I

// each thread corresponds to a particular column

// perform division on row to turn leading nonzero into a 1
// perform elimination on all other rows to make pivot column 0s
// call so Ax = AY and Ay = 2 * AX
__global__ void MatrixInverse(double *A, int Ax, int Ay) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double mult;
	double to_mult;
	double old_val;
	int current_pivot_col = 0;
	int i = 0;
	for (i = 0; i < Ax; i++) {
		// SWAP CODE
		if (i == col && A[i*Ay + col] == 0) {
			for (int k = i; k < Ax; k++) {
				if (A[k*Ay + col] != 0) {
					for (int x = 0; x < Ay; x++) {
						int tmp = A[i*Ay + x];
						A[i*Ay + x] = A[k*Ay + x];
						A[k*Ay + x] = tmp;
					}
					break;
				}

			}
		}

		// divide element by pivot
		__syncthreads();
		A[i*Ay + col] = A[i*Ay + col] / A[i*Ay + i];
		__syncthreads();

		for (int j = 0; j < Ax; j++) {
			mult = A[j*Ay + i];
			to_mult = A[i*Ay + col];
			old_val = A[j*Ay + col];
			if ((j != i) && (A[j*Ay + i] != 0)) {
				A[j*Ay + col] = old_val - mult * to_mult;
			}
		}

		__syncthreads();
	}
}

// Function that appends an identity matrix to the right of the current matrix
// keeping new matrix in row major form
// constant time in parallel
// assume that dst has 2*N*N = 2*len(src) allocated
__global__ void MatrixAppendIdentity(double* src, double* dst, int num_row, int num_col) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i % (2 * num_col) < num_col) {
		dst[i] = src[(num_row*(i / (2 * num_row))) + (i % (2 * num_row))];
	}
	else if ((i % (2 * num_row) - num_row == i / (2 * num_row))) {
		dst[i] = 1;
	}
	else {
		dst[i] = 0;
	}

}



__global__ void ExtractInverse(double *src, double* dst, int num_row, int num_col){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i % (2*num_col) >= num_col){
		dst[(num_row*(i / (2 * num_row))) + (i % (2 * num_row) - num_row)] = src[i];
    }


}







